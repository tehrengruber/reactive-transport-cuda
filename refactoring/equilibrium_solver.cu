#include <memory>
#include <map>
#include <string>
#include <hipblas.h>

#include "equilibrium_solver/equilibrium_solver_cuda.h"

#include "simple_cuda_profiler.h"
#include "equilibrium_solver.cpp"
#include "equilibrium_solver/cuda_equilibrium_solver.h"
#include "equilibrium_solver/minimization_result_info_cuda.h"
#include "kernel/equilibrate_cuda_kernel.cu"

#include "simple_timer.h"

namespace equilibrium_solver {

std::vector<MinimizationResultInfo> equilibrate_batch_cuda(ThermodynamicProperties& thermo_props,
                                                           Eigen::Matrix<numeric_t, Eigen::Dynamic, common::num_components, Eigen::RowMajor>& bs,
                                                           EquilibriumStateSOA& states,
                                                           MinimizerOptions options, bool init) {
    assert(init==false);
    size_t ncells = states.size();
    CudaBatchEquilibrationSolver solver(ncells, options);
    auto results = solver.equilibrate(thermo_props, bs, states);

    return results;
}

std::vector<MinimizationResultInfo> equilibrate_batch_cuda_v1(ThermodynamicProperties& thermo_props,
        Eigen::Matrix<numeric_t, Eigen::Dynamic, common::num_components, Eigen::RowMajor>& bs,
        std::vector<EquilibriumState>& states,
        MinimizerOptions options, bool init) {
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t ncells = states.size();
    std::vector<MinimizationResultInfo> results;
    results.resize(ncells);

    numeric_t* device_bs_ptr;
    EquilibriumState* device_states_ptr;
    MinimizationResultInfo* device_results_ptr;

    // allocate device memory
    if (hipMalloc((void**) &device_bs_ptr, ncells*num_components*sizeof(numeric_t)) != hipSuccess)
            throw std::runtime_error("Device memory allocation failed");
    if (hipMalloc((void**) &device_states_ptr, ncells*sizeof(EquilibriumState)) != hipSuccess)
            throw std::runtime_error("Device memory allocation failed");
    if (hipMalloc((void**) &device_results_ptr, ncells*sizeof(MinimizationResultInfo)) != hipSuccess)
        throw std::runtime_error("Device memory allocation failed");

    // copy data to device
    gpuErrchk( hipMemcpy((void*) device_bs_ptr, (void*) bs.data(), ncells*num_components*sizeof(numeric_t), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy((void*) device_states_ptr, (void*) states.data(), ncells*sizeof(EquilibriumState), hipMemcpyHostToDevice) );

    // compute block size and grid size
    int blockSize;
    int minGridSize;
    int gridSize;
    hipOccupancyMaxPotentialBlockSize(
            &minGridSize,
            &blockSize,
            (void*)equilibrate_cuda_kernel,
            0,
            ncells);
    gridSize = (ncells + blockSize - 1) / blockSize;

    {
        int numBlocks;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocks,
            (void*) equilibrate_cuda_kernel,
            blockSize,
            0);

        int device;
        hipDeviceProp_t prop;
        hipGetDevice(&device);
        hipGetDeviceProperties(&prop, device);

        std::cout << "Occupancy:" << double(numBlocks * blockSize)/prop.maxThreadsPerMultiProcessor << std::endl;
    }

    // call actual kernel
    hipEventRecord(start);
    equilibrate_cuda_kernel<<<gridSize, blockSize>>>(thermo_props, ncells, device_bs_ptr, device_states_ptr, options, init, device_results_ptr);
    hipEventRecord(stop);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // copy data back to host
    gpuErrchk(hipMemcpy((void*) states.data(), (void*) device_states_ptr, ncells*sizeof(EquilibriumState), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy((void*) results.data(), (void*) device_results_ptr, ncells*sizeof(MinimizationResultInfo), hipMemcpyDeviceToHost));

    // free memory
    if (hipFree((void*) device_bs_ptr) != hipSuccess)
            throw std::runtime_error("Device memory deallocation failed");
    if (hipFree((void*) device_states_ptr) != hipSuccess)
            throw std::runtime_error("Device memory deallocation failed");
    if (hipFree((void*) device_results_ptr) != hipSuccess)
        throw std::runtime_error("Device memory deallocation failed");

    // output throughput in processed cells/second
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    /*std::cout << "processed cells/s: " << 1000*ncells/milliseconds << std::endl;

    size_t total_iterations=0;
    for (size_t i=0; i<ncells; ++i) {
        total_iterations += results[i].it;
    }
    std::cout << "iterations/s: " << 1000*total_iterations/milliseconds << std::endl;*/

    return results;
}

}