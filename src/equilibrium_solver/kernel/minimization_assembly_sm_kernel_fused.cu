#include "hip/hip_runtime.h"
#include "common.h"
#include "common_cuda.h"
#include "chemistry.h"
#include "equilibrium_solver/minimizer_options.h"
#include "equilibrium_solver/equilibrium_state.h"
#include "equilibrium_solver/minimization_result_info_cuda.h"

namespace equilibrium_solver {

// make variables on the device visible
#ifdef __CUDA_ARCH__
using namespace common_device;
#else
using namespace common;
#endif

using chemistry::ThermodynamicProperties;

__global__
void minimization_assembly_sm_kernel_fused(ThermodynamicProperties thermo_props,
                                  MinimizerOptions options,
                                  MinimizationResultInfoCuda& info,
                                  Vector<numeric_t, common::num_species>* xs,
                                  Vector<numeric_t, common::num_components>* ys,
                                  Vector<numeric_t, common::num_species>* zs,
                                  numeric_t* bs_ptr,
                                  numeric_t* Js,
                                  numeric_t* Fs) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    constexpr size_t m = formula_matrix_t::RowsAtCompileTime;
    constexpr size_t n = formula_matrix_t::ColsAtCompileTime;
    constexpr size_t p = m + n;
    constexpr size_t t = m + 2 * n;
    const numeric_t mu = options.mu;

    __shared__ numeric_t values[16*(2*common::num_species+2*common::num_components+t)];

    if (idx < info.n_active) {
        int cidx = info.active_indices[idx];

        // initialize variables
        Map<Vector<numeric_t, common::num_species>, 0, InnerStride<16>> x(values+threadIdx.x);
        Map<Vector<numeric_t, common::num_components>, 0, InnerStride<16>> y(values+16*common::num_species+threadIdx.x);
        Map<Vector<numeric_t, common::num_species>, 0, InnerStride<16>> z(values+16*(common::num_species+common::num_components)+threadIdx.x);
        Map<component_amounts_t, 0, InnerStride<16>> b(values+16*(2*common::num_species+common::num_components)+threadIdx.x);
        Map<Vector<numeric_t, t>, 0, InnerStride<16>> F(values+16*(2*common::num_species+2*common::num_components)+threadIdx.x);

        // load data
        x = xs[cidx];
        y = ys[cidx];
        z = zs[cidx];
        b = Map<component_amounts_t>(bs_ptr+cidx*num_components);

        auto& A = formula_matrix;

        // assemble vector
        gibbs_energy_opt_inplace_gradient(thermo_props, x, F.head(n));
        F.head(n) -=  A.transpose()*y + z;
        F.segment(n, m) = A*x - b;
        F.tail(n) = (x.array() * z.array()).matrix() - mu*Vector<numeric_t, common::num_species>::Ones();
        F = -F;

        // write back
        Map<Vector<numeric_t, t>>(Fs+cidx*t) = F;

        // calculate error
        numeric_t error = F.template lpNorm<Infinity>();
        info.error[cidx] = error;
        info.converged[cidx] = error < options.tol;

        // assemble matrix
        Map<Matrix<numeric_t, t, t>> J(Js+cidx*t*t);
        if (!info.converged[cidx]) {
            J.setConstant(0);
            gibbs_energy_opt_inplace_hessian(thermo_props, x, J.block(0,   0,   n,   n));
            J.block(0,   n,   n,   p-n) = -A.transpose();
            J.block(0,   t-n, n,   n).diagonal().setConstant(-1);
            J.block(n,   0,   p-n, n) = A;
            J.block(t-n, 0,   n,   n).diagonal() = z;
            J.block(t-n, t-n, n,   n).diagonal() = x;
            ++info.iterations[cidx];
        }
    }
}

}